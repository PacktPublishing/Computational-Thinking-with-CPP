#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>

#include <thrust/universal_vector.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/execution_policy.h>

#include "hip/hip_cooperative_groups.h"

namespace cg = cooperative_groups;

__global__ void compute_2norm(const float* vectors,
                              float* norms,
                              unsigned n_vectors)
{
    constexpr unsigned vector_size = 32;

    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<32>(block);

    // For simplicity, assume that a block is 32 threads
    const unsigned my_vector_offset = grid.block_rank()*vector_size;

    float my_value = vectors[my_vector_offset + warp.thread_rank()];
    float my_norm = my_value * my_value;

    for (int offset=16; offset > 0; offset >>= 1) {
        my_norm += warp.shfl_down(my_norm, offset);
    }

    // only thread 0 writes to memory
    if (threadIdx.x == 0) {
        norms[blockIdx.x] = sqrtf(my_norm);
    }
}


int main() {
    auto num_vectors = 256;
    thrust::universal_vector<float> vectors(num_vectors * 32);
    thrust::universal_vector<float> norms(num_vectors);

    thrust::random::default_random_engine rng;
    thrust::random::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    thrust::generate(thrust::host, vectors.begin(), vectors.end(), [&] { return dist(rng); });

    compute_2norm<<<num_vectors, 32>>>(
        raw_pointer_cast(vectors.data()),
        raw_pointer_cast(norms.data()),
        num_vectors
        );
    hipDeviceSynchronize();

    std::cout << "Norms (" << norms.size() << "): \n";
    for (const auto& norm : norms) {
        std::cout << norm << "\n";
    }
    std::cout << "Done\n";

}